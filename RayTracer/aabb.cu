#include "hip/hip_runtime.h"
﻿#include "aabb.cuh"

namespace ray_tracer {

RT_DEVICE AABB::AABB(const Point3& min, const Point3& max) :
    minimum_(min), maximum_(max)
{
}

RT_DEVICE const Point3& AABB::min() const
{
    return minimum_;
}

RT_DEVICE const Point3& AABB::max() const
{
    return maximum_;
}

RT_DEVICE bool AABB::hit(const Ray& ray, float t_min, float t_max) const
{
    for (size_t dim = 0; dim < 3; ++dim)
    {
        auto t_near = (minimum_[dim] - ray.origin()[dim]) * ray.direction_inverse()[dim];
        auto t_far = (maximum_[dim] - ray.origin()[dim]) * ray.direction_inverse()[dim];

        if (ray.sign(dim))
        {
            const auto swap_val = t_near;
            t_near = t_far;
            t_far = swap_val;
        }

        t_min = std::fmaxf(t_min, t_near);
        t_max = std::fminf(t_max, t_far);

        if (t_max <= t_min)
        {
            return false;
        }
    }

    return true;
}

} // namespace ray_tracer
