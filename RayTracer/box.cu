﻿// --- Internal Includes ---
#include "box.cuh"
#include "solid_texture.cuh"
#include "xy_rect.cuh"
#include "yz_rect.cuh"
#include "xz_rect.cuh"

namespace ray_tracer {

RT_DEVICE Box::Box(const Point3& box_min, const Point3& box_max, Material* material) :
    box_min_(box_min), box_max_(box_max)
{
    material_ = material;
    const auto d_list = new Hittable * [6];

    d_list[0] = new XYRect(box_min_.x(), box_max_.x(), box_min_.y(), box_max_.y(), box_min_.z(), material_);
    d_list[1] = new XYRect(box_min_.x(), box_max_.x(), box_min_.y(), box_max_.y(), box_max_.z(), material_);

    d_list[2] = new YZRect(box_min_.y(), box_max_.y(), box_min_.z(), box_max_.z(), box_min_.x(), material_);
    d_list[3] = new YZRect(box_min_.y(), box_max_.y(), box_min_.z(), box_max_.z(), box_max_.x(), material_);

    d_list[4] = new XZRect(box_min_.x(), box_max_.x(), box_min_.z(), box_max_.z(), box_min_.y(), material_);
    d_list[5] = new XZRect(box_min_.x(), box_max_.x(), box_min_.z(), box_max_.z(), box_max_.y(), material_);

    sides_ = new HittableList(d_list, 6);
}

RT_DEVICE Box::~Box()
{
    // TODO: Implement CUDA compatible shared pointer
    // Set to nullptr to prevent multiple free of the same resource
    for(size_t index = 0; index < 6; ++index)
    {
        sides_->objects()[index]->material_ = nullptr;
    }

    delete sides_;
}

RT_DEVICE bool Box::hit(const Ray& ray, const float t_min, const float t_max, HitRecord& rec) const
{
    return sides_->hit(ray, t_min, t_max, rec);
}

RT_DEVICE bool Box::bounding_box(float ti, float tf, AABB& box_out) const
{
    box_out = AABB{ box_min_, box_max_ };
    return true;
}

} // namespace ray_tracer
