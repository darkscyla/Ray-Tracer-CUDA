﻿// --- Internal Includes ---
#include "bvh.cuh"
#include "empty.cuh"
#include "aabb_utils.cuh"

// --- Standard Includes ---
#include <thrust/sort.h>

namespace ray_tracer {

RT_DEVICE BVH::BVH(HittableList* hittable_list, hiprandState_t* rand_state, 
    const float ti, const float tf) :
    BVH(hittable_list->objects(), hittable_list->size(), rand_state, ti, tf)
{
}

RT_DEVICE BVH::BVH(Hittable** hittable_objects, const size_t length,
    hiprandState_t* rand_state, float ti, float tf) :
    hittable_objects_(hittable_objects)
{
    if (length == 1)
    {
        left_ = hittable_objects_[0];
        right_ = new Empty();
    }

    else if (length == 2)
    {
        left_ = hittable_objects_[0];
        right_ = hittable_objects_[1];
    }
    else
    {
        // Copy over the pointers
        const auto objects = new Hittable * [length];

        for (size_t index = 0; index < length; ++index)
        {
            objects[index] = hittable_objects_[index];
        }

        // We sort along a random direction, not the best but should work fine for now
        const auto axis = static_cast<size_t>(uniform_rand(rand_state, 0, 3)) % 3;
        thrust::sort(objects, objects + length, 
        [axis, ti, tf] RT_DEVICE(Hittable * a, Hittable * b) -> bool
            {
                return box_compare(a, b, axis, ti, tf);
            }
        );

        const auto mid = length / 2;

        left_ = new BVH(objects, mid, rand_state, ti, tf);
        right_ = new BVH(objects + mid, length - mid, rand_state, ti, tf);

        delete[] objects;
    }

    AABB box_left;
    AABB box_right;

    const auto left_has_bb = left_->bounding_box(ti, tf, box_left);
    const auto right_has_bb = right_->bounding_box(ti, tf, box_right);

    if (left_has_bb && right_has_bb)
    {
        box_ = enclosing_box(box_left, box_right);
    }
    else if(left_has_bb)
    {
        box_ = box_left;
    }
    else if(right_has_bb)
    {
        box_ = box_right;
    }
    else
    {
        printf("WARNING: Bounding box is not present for both the objects, this should never happen");
        has_bounding_box_ = false;
    }
}

RT_DEVICE BVH::~BVH()
{
    delete left_;
    delete right_;

    delete[] hittable_objects_;
}

RT_DEVICE bool BVH::hit(const Ray& ray, const float t_min, const float t_max, HitRecord& rec) const
{
    if (has_bounding_box_)
    {
        if (!box_.hit(ray, t_min, t_max))
        {
            return false;
        }
    }

    const auto left_hit = left_->hit(ray, t_min, t_max, rec);
    const auto right_hit = right_->hit(ray, t_min, left_hit ? rec.t : t_max, rec);

    return left_hit || right_hit;
}

RT_DEVICE bool BVH::bounding_box(const float ti, const float tf, AABB& box_out) const
{
    box_out = box_;
    return true;
}

} // namespace ray_tracer
