#include "hip/hip_runtime.h"
﻿#include "camera.cuh"

namespace ray_tracer {

RT_DEVICE Camera::Camera(const Vec3& background, const size_t depth, 
    const Point3& look_from, const Point3& look_at, const Vec3& up_vector,
    const float v_fov, const float aspect_ratio, const float aperture,
    const float focus_distance, const float ti, const float tf) :
        background_(background),
        depth_(depth)
{
    const auto theta = deg_to_rad(v_fov);
    const auto h = std::tan(theta / 2);
    const auto viewport_height = 2.0f * h;
    const auto viewport_width = aspect_ratio * viewport_height;

    // Aligned camera directions
    w_ = unit_vector(look_from - look_at);
    u_ = unit_vector(cross(up_vector, w_));
    v_ = unit_vector(cross(w_, u_));

    origin_ = look_from;
    horizontal_ = focus_distance * viewport_width * u_;
    vertical_ = focus_distance * viewport_height * v_;
    lower_left_corner_ = origin_ - horizontal_ / 2 - vertical_ / 2 - focus_distance * w_;

    lens_radius_ = aperture / 2;

    ti_ = ti;
    tf_ = tf;
}

RT_DEVICE Ray Camera::get_ray(hiprandState_t* rand_state, const float u, const float v) const
{
    const auto random_direction = lens_radius_ * random_unit_planer(rand_state);
    const auto offset = u_ * random_direction.x() + v_ * random_direction.y();
    const auto offset_origin = origin_ + offset;

    return { offset_origin, lower_left_corner_ + u * horizontal_ + v * vertical_ - offset_origin,
        uniform_rand(rand_state, ti_, tf_)
    };
}

RT_DEVICE const Color& Camera::background() const
{
    return background_;
}

RT_DEVICE size_t Camera::depth() const
{
    return depth_;
}

} // namespace ray_tracer
