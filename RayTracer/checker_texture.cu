#include "hip/hip_runtime.h"
﻿// --- Internal Includes ---
#include "checker_texture.cuh"
#include "solid_texture.cuh"

// --- Standard Includes ---
#include <cmath>

namespace ray_tracer {

RT_DEVICE CheckerTexture::CheckerTexture(Texture* even, Texture* odd) :
    even_(even), odd_(odd)
{
}

RT_DEVICE CheckerTexture::CheckerTexture(const Color& even_color, const Color& odd_color) :
    even_(new SolidTexture(even_color)),
    odd_(new SolidTexture(odd_color))
{
}

RT_DEVICE CheckerTexture::~CheckerTexture()
{
    delete even_;
    delete odd_;
}

RT_DEVICE Color CheckerTexture::value(const Point3& point, const float u, const float v) const
{
    //const auto sins_sign = ( std::sin(10 * point.x()) * 
    //                         std::sin(10 * point.y()) * 
    //                         std::sin(10 * point.z()) ) < 0;
    const auto sins_sign = (std::sin(500 * u) * std::sin(500 * v) < 0);

    return sins_sign ? odd_->value(point, u, v) : even_->value(point, u, v);
}

} // namespace ray_tracer
