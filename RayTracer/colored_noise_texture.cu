﻿#include "colored_noise_texture.cuh"
#include "solid_texture.cuh"

namespace ray_tracer {

RT_DEVICE ColoredNoiseTexture::ColoredNoiseTexture(const float scale, Texture* low, Texture* high,
    hiprandState_t* rand_state, const size_t depth, const NoiseFunction function):
    NoiseTexture(scale, rand_state, depth, function),
    low_(low), high_(high)
{
}

RT_DEVICE ColoredNoiseTexture::ColoredNoiseTexture(const float scale, const Color& low, const Color& high, 
    hiprandState_t* rand_state, const size_t depth, const NoiseFunction function) :
    ColoredNoiseTexture(scale, new SolidTexture(low), new SolidTexture(high), rand_state, 
        depth, function)
{
}

RT_DEVICE ColoredNoiseTexture::~ColoredNoiseTexture()
{
    delete low_;
    delete high_;
}

RT_DEVICE Color ColoredNoiseTexture::value(const Point3& point, float u, float v) const
{
    const auto t = function_(scale_, point, perlin_noise_, depth_);
    return (1 - t) * low_->value(point, u, v) + t * high_->value(point, u, v);
}

} // namespace ray_tracer
