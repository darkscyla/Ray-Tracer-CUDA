#include "hip/hip_runtime.h"
﻿#include "constant_volume.cuh"
#include "isotropic.cuh"

namespace ray_tracer {

RT_DEVICE ConstantVolume::ConstantVolume(Hittable* hittable, const float density, const Color& color) :
    ConstantVolume(hittable, density, new Isotropic(color))
{
}

RT_DEVICE ConstantVolume::ConstantVolume(Hittable* hittable, const float density, Material* material) :
    hittable_(hittable), neg_density_inv_(-1.0f / density)
{
    material_ = material;
    hiprand_init(786, 0, 0, &rand_state_);
}

RT_DEVICE ConstantVolume::~ConstantVolume()
{
    delete hittable_;
}

RT_DEVICE bool ConstantVolume::hit(const Ray& ray, const float t_min, const float t_max, HitRecord& rec) const
{
    // The idea here is to make sure that the underlying object indeed has a volume.
    // For that, we check for 2 intersections with the object. This of course would
    // not hold if the object has holes in it or the object is non-convex
    HitRecord lower;
    HitRecord upper;

    if (!hittable_->hit(ray, -kInfinity, kInfinity, lower))
    {
        return false;
    }

    static const auto eps = 1e-4f;
    // We move the point a bit further to avoid getting the same point again
    if (!hittable_->hit(ray, lower.t + eps, kInfinity, upper))
    {
        return false;
    }

    if (t_min > lower.t) lower.t = t_min;
    if (t_max < upper.t) upper.t = t_max;

    if (lower.t >= upper.t)
    {
        return false;
    }

    if(lower.t < 0.0f) lower.t = 0.0f;

    const auto ray_length = ray.direction().length();
    const auto distance_inside_volume = (upper.t - lower.t) * ray_length;
    const auto random_scatter_distance = neg_density_inv_ * std::log(random_unit(&rand_state_));

    if (random_scatter_distance > distance_inside_volume)
    {
        return false;
    }

    rec.t = lower.t + random_scatter_distance / ray_length;
    rec.hit_point = ray.at(rec.t);

    rec.normal = lower.normal;
    rec.front_face = lower.front_face;
    rec.material = material_ ? material_ : lower.material;

    return true;
}

RT_DEVICE bool ConstantVolume::bounding_box(const float ti, const float tf, AABB& box_out) const
{
    return hittable_->bounding_box(ti, tf, box_out);
}

} // namespace rey_tracer
