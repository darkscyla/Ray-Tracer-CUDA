#include "hip/hip_runtime.h"
﻿// --- Internal Includes ---
#include "dielectric.cuh"
#include "hittable.cuh"

namespace ray_tracer {

RT_DEVICE Dielectric::Dielectric(const Color& color, const float refractive_index) :
    albedo_(color), refractive_index_(refractive_index)
{
}

RT_DEVICE bool Dielectric::scatter(hiprandState_t* rand_state, const Ray& ray_incident,
    const HitRecord& rec, Color& attenuation, Ray& ray_scattered) const
{
    const auto refractive_index = rec.front_face ? 1 / refractive_index_ : refractive_index_;
    const auto unit_incident_ray_direction = unit_vector(ray_incident.direction());

    // Incident ray angle calculations
    const auto cos_incident = -dot(unit_incident_ray_direction, rec.normal);
    const auto sin_incident = std::sqrt(std::abs(1 - cos_incident * cos_incident));

    Vec3 scatter_direction;

    // In case refraction is not possible or desired
    if ((refractive_index * sin_incident > 1.0f) || 
        reflectance(cos_incident, refractive_index) > random_unit(rand_state))
    {
        scatter_direction = reflect(unit_incident_ray_direction, rec.normal);
    }
    else
    {
        scatter_direction = refract(unit_incident_ray_direction, cos_incident, rec.normal, refractive_index);
    }

    ray_scattered = Ray(rec.hit_point, scatter_direction, ray_incident.time());
    attenuation = albedo_;

    return true;
}

RT_DEVICE float Dielectric::reflectance(const float cos_incident, const float refractive_index)
{
    auto r0 = (1 - refractive_index) / (1 + refractive_index);
    r0 *= r0;

    return r0 + (1 - r0) * std::powf(1 - cos_incident, 5);
}

} // namespace ray_tracer
