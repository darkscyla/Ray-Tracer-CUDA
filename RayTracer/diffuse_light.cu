﻿#include "diffuse_light.cuh"
#include "solid_texture.cuh"

namespace ray_tracer {

RT_DEVICE DiffuseLight::DiffuseLight(Texture* texture, const float emissivity) :
    emit_(texture),
    emissivity_(emissivity)
{
}

RT_DEVICE DiffuseLight::DiffuseLight(const Color& color, const float emissivity) :
    DiffuseLight(new SolidTexture(color), emissivity)
{
}

RT_DEVICE DiffuseLight::~DiffuseLight()
{
    delete emit_;
}

RT_DEVICE bool DiffuseLight::scatter(hiprandState_t* rand_state, const Ray& ray_incident,
    const HitRecord& rec, Color& attenuation, Ray& ray_scattered) const
{
    return false;
}

RT_DEVICE Color DiffuseLight::emitted(const Point3& point, const float u, const float v)
{
    return emissivity_ * emit_->value(point, u, v);
}

} // namespace ray_tracer
