﻿#include "empty.cuh"

namespace ray_tracer {

RT_DEVICE Empty::Empty()
{
}

RT_DEVICE bool Empty::hit(const Ray& ray, float t_min, float t_max, HitRecord& rec) const
{
    return false;
}

RT_DEVICE bool Empty::bounding_box(float ti, float tf, AABB& box_out) const
{
    return false;
}

} // namespace ray_tracer
