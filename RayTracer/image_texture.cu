﻿#include "image_texture.cuh"

namespace ray_tracer {

RT_DEVICE ImageTexture::ImageTexture(ImageWrapper* image) :
    image_(image)
{
}

RT_DEVICE Color ImageTexture::value(const Point3& point, float u, float v) const
{
    if(image_->rows() == 0 || image_->cols() == 0)
    {
        return { 0.0f, 1.0f, 1.0f };
    }

    u = clamp(u, 0.0, 1.0);
    v = 1 - clamp(v, 0.0, 1.0); // Flip coordinates as OpenCV indexes images oppositely

    auto i = static_cast<size_t>(u * image_->cols());
    auto j = static_cast<size_t>(v * image_->rows());

    // The limits of u and v should be [0, 1)
    if (i >= image_->cols())
    {
        i = image_->cols() - 1;
    }
    if (j >= image_->rows())
    {
        j = image_->rows() - 1;
    }

    return image_->at(j, i);
}

} // namespace ray_tracer
