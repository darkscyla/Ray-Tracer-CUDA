﻿#include "isotropic.cuh"
#include "solid_texture.cuh"
#include "hittable.cuh"

namespace ray_tracer {

RT_DEVICE Isotropic::Isotropic(const Color& color) :
    Isotropic(new SolidTexture(color))
{
}

RT_DEVICE Isotropic::Isotropic(Texture* texture) :
    albedo_(texture)
{
}

RT_DEVICE Isotropic::~Isotropic()
{
    delete albedo_;
}

RT_DEVICE bool Isotropic::scatter(hiprandState_t* rand_state, const Ray& ray_incident, const HitRecord& rec, Color& attenuation,
    Ray& ray_scattered) const
{
    ray_scattered = Ray(rec.hit_point, random_unit_vector(rand_state), ray_incident.time());
    attenuation = albedo_->value(rec.hit_point, rec.u, rec.v);
    return true;
}

} // namespace ray_tracer
