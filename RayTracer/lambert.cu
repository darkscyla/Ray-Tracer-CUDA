﻿// --- Internal Includes ---
#include "lambert.cuh"
#include "hittable.cuh"
#include "solid_texture.cuh"

namespace ray_tracer {

RT_DEVICE Lambert::Lambert(const Color& color) : albedo_( new SolidTexture(color))
{
}

RT_DEVICE Lambert::Lambert(Texture* texture): albedo_(texture)
{
}

RT_DEVICE Lambert::~Lambert()
{
    delete albedo_;
}

RT_DEVICE bool Lambert::scatter(hiprandState_t* rand_state, const Ray& ray_incident,
                      const HitRecord& rec, Color& attenuation, Ray& ray_scattered) const
{
    ray_scattered = Ray(rec.hit_point, random_aligned_unit_vector(rand_state, rec.normal), ray_incident.time());
    attenuation = albedo_->value(rec.hit_point, rec.u, rec.v) ;

    return true;
}

} // namespace ray_tracer
