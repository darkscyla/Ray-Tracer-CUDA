#include "hip/hip_runtime.h"
﻿// --- Internal Includes  ---
#include "rt_utils.cuh"
#include "vec3.cuh"
#include "Ray.cuh"
#include "hittable.cuh"
#include "camera.cuh"
#include "scene_selector.cuh"
#include "image_wrapper.cuh"

// --- CUDA Includes ---
#include <>
#include <hiprand/hiprand_kernel.h>

// --- OpenCV Includes ---
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

// --- Parallel Includes ---
#include <omp.h>

using namespace ray_tracer;

RT_DEVICE Color color(const Ray& ray, const Color& background, Hittable** world, const size_t depth, hiprandState_t* rand_state)
{
    auto cur_ray = ray;
    auto cur_attenuation = Vec3{ 1.0f, 1.0f, 1.0f };
    auto gathered_light = Color({0.0f, 0.0f, 0.0f});
    HitRecord rec;

    for (size_t i = 0; i < depth; ++i)
    {
        if ((*world)->hit(cur_ray, 0.001f, kInfinity, rec))
        {
            Ray scattered;
            Color attenuation;
            const auto emitted = rec.material->emitted(rec.hit_point, rec.u, rec.v);

            if (rec.material->scatter(rand_state, cur_ray, rec, attenuation, scattered))
            {
                gathered_light += emitted * cur_attenuation;
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else
            {
                return gathered_light + (emitted * cur_attenuation);
            }
        }
        else
        {
            return gathered_light + (background * cur_attenuation);
        }
    }

    return { 0.0f, 0.0f, 0.0f };
}

RT_GLOBAL void render(Vec3* fb, const size_t max_x, const size_t max_y,
    const size_t samples, Camera** camera, Hittable** world, hiprandState_t* rand_state)
{
    const auto i = threadIdx.x + blockIdx.x * blockDim.x;
    const auto j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= max_x || j >= max_y)
    {
        return;
    }

    const auto pixel_index = j * max_x + i;
    auto local_rand_state = rand_state[pixel_index];

    Vec3 col{ 0.0f, 0.0f, 0.0f };

    for (size_t k = 0; k < samples; ++k)
    {
        const auto u = static_cast<float>(i + hiprand_uniform(&local_rand_state)) / max_x;
        const auto v = static_cast<float>(j + hiprand_uniform(&local_rand_state)) / max_y;

        col += color((*camera)->get_ray(&local_rand_state, u, v), 
            (*camera)->background(), world, (*camera)->depth(), &local_rand_state);
    }

    const auto scale = 1.0f / static_cast<float>(samples);

    fb[pixel_index] = Color {
        std::cbrt(scale * col[0]),
        std::cbrt(scale * col[1]),
        std::cbrt(scale * col[2])
    };
    color_correct(fb[pixel_index]);
}

RT_GLOBAL void destroy_world(Hittable** d_world, Camera** d_camera)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        delete* (d_world);
        delete* (d_camera);
    }
}

RT_GLOBAL void setup_rand(hiprandState_t* rand_state, const size_t max_x, const size_t max_y)
{
    const auto i = threadIdx.x + blockIdx.x * blockDim.x;
    const auto j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i >= max_x || j >= max_y)
    {
        return;
    }

    const auto pixel_index = j * max_x + i;

    hiprand_init(1984 + pixel_index, 0, 0, &rand_state[pixel_index]);
}

int main()
{
    //constexpr auto aspect_ratio = 1.0;
    constexpr auto aspect_ratio = 16.0f / 9.0f;
    constexpr size_t image_height = 1080;
    //constexpr size_t image_height = 10;
    constexpr auto image_width = static_cast<size_t>(image_height * aspect_ratio);
    constexpr size_t samples_per_pixel = 10;
    const size_t scene_id = 7;

    const auto pixels = image_width * image_height;

    Vec3* vb;
    CUDA_CHECK_ERRORS(hipMallocManaged(&vb, pixels * sizeof(Vec3)));

    // Create the world and camera
    Hittable** d_world;
    CUDA_CHECK_ERRORS(hipMalloc(&d_world, sizeof(Hittable*)));

    Camera** d_camera;
    CUDA_CHECK_ERRORS(hipMalloc(&d_camera, sizeof(Camera*)));

    const size_t num_textures = 2;
    ImageWrapper h_textures[num_textures] = {
        ImageWrapper("resources/earth.jpg"),
        ImageWrapper("resources/abs_final.png")
    };

    ImageWrapper* d_textures;
    CUDA_CHECK_ERRORS(hipMalloc(&d_textures, num_textures * sizeof(ImageWrapper)));
    hipMemcpy(d_textures, h_textures, num_textures * sizeof(ImageWrapper), hipMemcpyHostToDevice);

    // For creating the world, we use BVH and need to increase the stack size
    size_t size;
    hipDeviceGetLimit(&size, hipLimitStackSize);
    hipDeviceSetLimit(hipLimitStackSize, 8 * size);

    select_scene<< <1, 1 >> >(scene_id, d_textures, d_world, d_camera, aspect_ratio);
    CUDA_CHECK_ERRORS(hipGetLastError());
    CUDA_CHECK_ERRORS(hipDeviceSynchronize());

    //// Restore the original stack size
    //hipDeviceSetLimit(hipLimitStackSize, size);

    const size_t tx = 16;
    const size_t ty = 16;

    dim3 blocks{ image_width / tx + 1, image_height / ty + 1 };
    dim3 threads{ tx, ty };

    // Setup the CUDA rand stuff
    hiprandState_t* d_rand_state;
    CUDA_CHECK_ERRORS(hipMalloc(&d_rand_state, pixels * sizeof(hiprandState_t)));

    std::cout << "CUDA: Generating random states" << std::endl;
    setup_rand << <blocks, threads >> > (d_rand_state, image_width, image_height);
    CUDA_CHECK_ERRORS(hipGetLastError());
    CUDA_CHECK_ERRORS(hipDeviceSynchronize());

    // Render the image
    const auto start_time = std::chrono::high_resolution_clock::now();

    std::cout << "CUDA: Rendering..." << std::endl;
    render << <blocks, threads >> > (vb, image_width, image_height, 
        samples_per_pixel, d_camera, d_world, d_rand_state);

    CUDA_CHECK_ERRORS(hipGetLastError());
    CUDA_CHECK_ERRORS(hipDeviceSynchronize());

    const auto time_elapsed = std::chrono::duration_cast<std::chrono::seconds>(
        std::chrono::high_resolution_clock::now() - start_time).count();
    std::cout << "GPU took: " << time_elapsed << " secs..." << std::endl;

    cv::Mat image = cv::Mat::zeros(image_height, image_width, CV_8UC3);

    #pragma omp parallel for
    for (auto i = 0; i < image_width; ++i)
    {
        for (auto j = 0; j < image_height; ++j)
        {
            // OpenCV uses BGR
            const auto pixel_index = j * image_width + i;
            image.at<cv::Vec3b>(cv::Point(static_cast<int>(i), static_cast<int>(j))) = cv::Vec3b{
                static_cast<uint8_t>(255.99 * vb[pixel_index][2]),
                static_cast<uint8_t>(255.99 * vb[pixel_index][1]),
                static_cast<uint8_t>(255.99 * vb[pixel_index][0])
            };
        }
    }

    cv::flip(image, image, 0);
    cv::imwrite("result.png", image);

    destroy_world << <1, 1 >> > (d_world, d_camera);
    CUDA_CHECK_ERRORS(hipGetLastError());
    CUDA_CHECK_ERRORS(hipDeviceSynchronize());

    CUDA_CHECK_ERRORS(hipFree(d_textures));
    CUDA_CHECK_ERRORS(hipFree(d_camera));
    CUDA_CHECK_ERRORS(hipFree(d_world));
    CUDA_CHECK_ERRORS(hipFree(d_rand_state));
    CUDA_CHECK_ERRORS(hipFree(vb));

    // Manually cleanup the textures that reside in device memory
    for(size_t index = 0; index < num_textures; ++index)
    {
        h_textures[index].release_device_data();
    }

    CUDA_CHECK_ERRORS(hipDeviceReset());

    return 0;
}
