﻿// --- Internal Includes ---
#include "metal.cuh"
#include "hittable.cuh"

namespace ray_tracer {

RT_DEVICE Metal::Metal(const Color& color, const float roughness) :
    albedo_(color), roughness_(clamp(roughness, 0.0f, 1.0f))
{
}

RT_DEVICE bool Metal::scatter(hiprandState_t* rand_state, const Ray& ray_incident, 
    const HitRecord& rec,Color& attenuation, Ray& ray_scattered) const
{
    const auto reflect_direction = reflect(unit_vector(ray_incident.direction()), rec.normal);
    ray_scattered = Ray(rec.hit_point, reflect_direction + 
        roughness_ * random_aligned_unit_vector(rand_state,  rec.normal), ray_incident.time());
    attenuation = albedo_;

    return true;
}

} // namespace ray_tracer
