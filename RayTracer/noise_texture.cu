#include "hip/hip_runtime.h"
﻿#include "noise_texture.cuh"

namespace ray_tracer {

RT_DEVICE NoiseTexture::NoiseTexture(float scale, hiprandState_t* rand_state, 
    const size_t depth, NoiseFunction function) :
        perlin_noise_(rand_state), scale_(scale), depth_(depth)
{
    function_ = function ? std::move(function) : []
    RT_DEVICE (const float scale, const Point3& point, const Perlin& perlin, const size_t depth) -> float
    {
        return 0.5f * (1 + std::sin(scale * point.z() + 10 * perlin.turbulence(point, depth)));
    };
}

RT_DEVICE Color NoiseTexture::value(const Point3& point, float u, float v) const
{
    return Color{1.0f, 1.0f, 1.0f} * function_(scale_, point, perlin_noise_, depth_);
}

} // namespace ray_tracer
