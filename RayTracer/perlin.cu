#include "hip/hip_runtime.h"
﻿// --- Internal Includes ---
#include "perlin.cuh"

// --- Standard Includes ---
#include <numeric>

namespace ray_tracer {

RT_DEVICE Perlin::Perlin(hiprandState_t* rand_state)
{
    for(size_t index = 0; index < lattice_points_; ++index)
    {
        random_vectors_[index] = random_unit_vector(rand_state);
    }

    perlin_generate_permutation(perm_x_, rand_state);
    perlin_generate_permutation(perm_y_, rand_state);
    perlin_generate_permutation(perm_z_, rand_state);
}

RT_DEVICE float Perlin::noise(const Point3& point) const
{
    const auto fx = std::floor(point.x());
    const auto fy = std::floor(point.y());
    const auto fz = std::floor(point.z());

    const auto i = static_cast<int>(fx);
    const auto j = static_cast<int>(fy);
    const auto k = static_cast<int>(fz);

    // Perform the Hermitian smoothing
    auto u = point.x() - fx;
    auto v = point.y() - fy;
    auto w = point.z() - fz;
    u = u * u * (3 - 2 * u);
    v = v * v * (3 - 2 * v);
    w = w * w * (3 - 2 * w);

    float accumulate = 0;

    // Pre-compute the products
    float dx_u[2] = {0, u};
    float dy_v[2] = { 0, v };
    float dz_w[2] = {0, w};

    // For powers of 2, & is equivalent to mod operator
    // Get the accumulate using tri-linear interpolation
    for (size_t dx = 0; dx < 2; ++dx)
    {
        for (size_t dy = 0; dy < 2; ++dy)
        {
            for (size_t dz = 0; dz < 2; ++dz)
            {
                const Vec3 weight = { u - dx, v - dy, w - dz };
                accumulate += (dx_u[dx] + (1 - dx) * (1 - u)) *
                    (dy_v[dy] + (1 - dy) * (1 - v)) *
                    (dz_w[dz] + (1 - dz) * (1 - w)) *
                    dot(random_vectors_[
                        perm_x_[(i + dx) & 255] ^
                            perm_y_[(j + dy) & 255] ^
                            perm_z_[(k + dz) & 255]
                    ], weight);
            }
        }
    }

    return accumulate;
}

RT_DEVICE float Perlin::turbulence(const Point3& point, const size_t depth) const
{
    auto accumulate = 0.0;
    auto scalable_point = point;
    auto weight = 1.0;

    for (size_t index = 0; index < depth; ++index)
    {
        accumulate += weight * noise(scalable_point);
        weight *= 0.5;
        scalable_point *= 2;
    }

    return std::fabs(accumulate);
}

RT_DEVICE void Perlin::perlin_generate_permutation(Cache<size_t>& perm, hiprandState_t* rand_state)
{
    for(size_t index = 0; index < lattice_points_; ++index)
    {
        perm[index] = index;
    }

    permute(perm, lattice_points_, rand_state);
}

RT_DEVICE void Perlin::permute(Cache<size_t>& perm, size_t n, hiprandState_t* rand_state)
{
    for (auto index = lattice_points_ - 1; index > 0; --index)
    {
        const auto rand_index = static_cast<size_t>(
            uniform_rand(rand_state, 0, lattice_points_)) % lattice_points_;
        
        // Swap the value of the lattice points
        const auto swap = perm[index];
        perm[index] = perm[rand_index];
        perm[rand_index] = swap;
    }
}

} // namespace ray_tracer
