﻿// --- Internal Includes ---
#include "ray.cuh"

namespace ray_tracer {

RT_DEVICE Ray::Ray(const Point3& origin, const Vec3& direction, const float time) :
    orig_{ origin }, dir_{ direction },
    inv_dir_{ 1 / direction.x(), 1 / direction.y(), 1 / direction.z() },
    sign_{ inv_dir_[0] < 0, inv_dir_[1] < 0, inv_dir_[2] < 0 },
    time_{ time }
{
}

RT_DEVICE const Point3& Ray::origin() const
{
    return orig_;
}

RT_DEVICE const Vec3& Ray::direction() const
{
    return dir_;
}

RT_DEVICE  const Vec3& Ray::direction_inverse() const
{
    return inv_dir_;
}

RT_DEVICE bool Ray::sign(const size_t direction) const
{
    return sign_[direction];
}

RT_DEVICE float Ray::time() const
{
    return time_;
}

RT_DEVICE Point3 Ray::at(const float t) const
{
    return orig_ + t * dir_;
}

} // namespace ray_tracer
