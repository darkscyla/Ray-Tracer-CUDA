#include "hip/hip_runtime.h"
﻿#include "rotate_y.cuh"

namespace ray_tracer {

RT_DEVICE RotateY::RotateY(Hittable* hittable, const float angle_deg) :
    hittable_(hittable)
{
    const auto radians = deg_to_rad(angle_deg);
    cos_theta_ = std::cos(radians);
    sin_theta_ = std::sin(radians);

    // Check if bounding box exists for the base object. We use arbitrary values
    has_box_ = hittable_->bounding_box(0.0f, 1.0f, bounding_box_);

    if (!has_box_)
    {
        return;
    }

    Point3 min{ kInfinity, bounding_box_.min().y(), kInfinity };
    Point3 max{ -kInfinity, bounding_box_.max().y(), -kInfinity };

    for (size_t dx = 0; dx < 2; ++dx)
    {
        for (size_t dz = 0; dz < 2; ++dz)
        {
            // We generate the permutation of the min and max x-z points
            const auto x = dx * bounding_box_.max().x() + (1 - dx) * bounding_box_.min().x();
            const auto z = dz * bounding_box_.max().z() + (1 - dz) * bounding_box_.min().z();

            // Rotation is about origin
            const auto x_dot = x * cos_theta_ + z * sin_theta_;
            const auto z_dot = -x * sin_theta_ + z * cos_theta_;

            const Vec3 extent{ x_dot, 0.0f, z_dot };

            for (size_t dim = 0; dim < 3; dim += 2)
            {
                min[dim] = std::fmin(min[dim], extent[dim]);
                max[dim] = std::fmax(max[dim], extent[dim]);
            }
        }
    }

    bounding_box_ = AABB{ min, max };
}

RT_DEVICE RotateY::~RotateY()
{
    delete hittable_;
}

RT_DEVICE bool RotateY::hit(const Ray& ray, const float t_min, const float t_max, HitRecord& rec) const
{
    auto origin = ray.origin();
    auto direction = ray.direction();

    // Rotate the ray by angle
    origin[0] = ray.origin().x() * cos_theta_ - ray.origin().z() * sin_theta_;
    origin[2] = ray.origin().x() * sin_theta_ + ray.origin().z() * cos_theta_;

    direction[0] = ray.direction().x() * cos_theta_ - ray.direction().z() * sin_theta_;
    direction[2] = ray.direction().x() * sin_theta_ + ray.direction().z() * cos_theta_;

    const Ray ray_rotated{ origin, direction, ray.time() };

    if (!hittable_->hit(ray_rotated, t_min, t_max, rec))
    {
        return false;
    }

    auto point = rec.hit_point;
    auto normal = rec.normal;

    // Un-rotate the ray by angle
    point[0] = rec.hit_point.x() * cos_theta_ + rec.hit_point.z() * sin_theta_;
    point[2] = -rec.hit_point.x() * sin_theta_ + rec.hit_point.z() * cos_theta_;

    normal[0] = rec.normal.x() * cos_theta_ + rec.normal.z() * sin_theta_;
    normal[2] = -rec.normal.x() * sin_theta_ + rec.normal.z() * cos_theta_;

    rec.hit_point = point;
    rec.set_face_normal(ray_rotated, normal);

    return true;
}

RT_DEVICE bool RotateY::bounding_box(float ti, float tf, AABB& box_out) const
{
    if (has_box_)
    {
        box_out = bounding_box_;
        return true;
    }

    return false;
}

} // namespace ray_tracer
