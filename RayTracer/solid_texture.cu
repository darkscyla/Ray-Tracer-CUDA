﻿#include "solid_texture.cuh"

namespace ray_tracer {

RT_DEVICE SolidTexture::SolidTexture(const Color& color) : color_(color)
{
}

RT_DEVICE SolidTexture::SolidTexture(const float r, const float g, const float b) : color_({ r, g, b })
{
}

RT_DEVICE Color SolidTexture::value(const Point3& point, const float u, const float v) const
{
    return color_;
}

} // namespace ray_tracer
