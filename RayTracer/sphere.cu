#include "hip/hip_runtime.h"
#include "sphere.cuh"

namespace ray_tracer {

RT_DEVICE Sphere::Sphere(const Point3& origin, const float radius, Material* material) :
    center_(origin), radius_(radius), radius_inv_(1.0f / radius), radius_2_(radius* radius)
{
    material_ = material;
}

RT_DEVICE bool Sphere::hit(const Ray& ray, const float t_min, const float t_max, HitRecord& rec) const
{
    const auto oc = ray.origin() - center_;
    const auto a = ray.direction().length_squared();
    const auto b_half = dot(ray.direction(), oc);
    const auto c = oc.length_squared() - radius_2_;
    const auto disc = b_half * b_half - a * c;

    if (disc < 0.0f)
        return false;

    const auto root = std::sqrt(disc);
    const auto a_inv = 1.0f / a;

    auto t = (-b_half - root) * a_inv;

    for (size_t i = 0; i < 2; ++i, t = (-b_half + root) * a_inv)
    {
        if (t < t_max && t > t_min)
        {
            rec.t = t;
            rec.hit_point = ray.at(t);
            const auto outward_normal = (rec.hit_point - center_) * radius_inv_;
            rec.set_face_normal(ray, outward_normal);
            get_uv_coordinates(outward_normal, rec.u, rec.v);
            rec.material = material_;

            return true;
        }
    }

    // Out of bounds
    return false;
}

RT_DEVICE bool Sphere::bounding_box(const float ti, const float tf, AABB& box_out) const
{
    box_out = AABB{
        center_ - Vec3{ radius_, radius_, radius_ },
        center_ + Vec3{ radius_, radius_, radius_ }
    };

    return true;
}

RT_DEVICE void Sphere::get_uv_coordinates(const Point3& point, float& u, float& v)
{
    const auto theta = std::acosf(-point.y());
    const auto phi = std::atan2f(-point.z(), point.x()) + kPi;

    u = phi * k1by2Pi;
    v = theta * k1byPi;
}

} // namespace ray_tracer
