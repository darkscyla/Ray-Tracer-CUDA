#include "hip/hip_runtime.h"
﻿// --- Internal Includes ---
#include "sphere_moving.cuh"
#include "aabb_utils.cuh"

namespace ray_tracer {

RT_DEVICE SphereMoving::SphereMoving(const Point3& origin_initial, const Point3& origin_final,
    const float ti, const float tf, const float radius, Material* material) :
        center_initial_(origin_initial), center_final_(origin_final), ti_(ti), dt_inv_(1.0f / (tf - ti)), radius_(radius),
        radius_inv_(1.0f / radius), radius_2_(radius* radius)
{
    material_ = material;
}

RT_DEVICE Point3 SphereMoving::center(const float time) const
{
    const auto t = (time - ti_) * dt_inv_;
    return (1 - t) * center_initial_ + t * center_final_;
}

RT_DEVICE bool SphereMoving::hit(const Ray& ray, const float t_min, const float t_max, HitRecord& rec) const
{
    const auto oc = ray.origin() - center(ray.time());
    const auto a = ray.direction().length_squared();
    const auto b_half = dot(ray.direction(), oc);
    const auto c = oc.length_squared() - radius_2_;
    const auto disc = b_half * b_half - a * c;

    // If no intersection, we return a negative number
    if (disc < 0.0)
        return false;

    const auto root = std::sqrt(disc);
    const auto a_inv = 1 / a;

    auto t = (-b_half - root) / a;

    for (size_t i = 0; i < 2; ++i, t = (-b_half + root) * a_inv)
    {
        if (t < t_max && t > t_min)
        {
            rec.t = t;
            rec.hit_point = ray.at(t);
            const auto outward_normal = (rec.hit_point - center(ray.time())) * radius_inv_;
            rec.set_face_normal(ray, outward_normal);
            get_uv_coordinates(outward_normal, rec.u, rec.v);
            rec.material = material_;

            return true;
        }
    }

    // Out of bounds
    return false;
}

RT_DEVICE bool SphereMoving::bounding_box(const float ti, const float tf, AABB& box_out) const
{
    const auto box_i = AABB(
        center(ti) - Vec3{ radius_, radius_, radius_ },
        center(ti) + Vec3{ radius_, radius_, radius_ }
    );

    const auto box_f = AABB(
        center(tf) - Vec3{ radius_, radius_, radius_ },
        center(tf) + Vec3{ radius_, radius_, radius_ }
    );

    box_out = enclosing_box(box_i, box_f);

    return true;
}

RT_DEVICE void SphereMoving::get_uv_coordinates(const Point3& point, float& u, float& v)
{
    const auto theta = std::acosf(-point.y());
    const auto phi = std::atan2f(-point.z(), point.x()) + kPi;

    u = phi * k1by2Pi;
    v = theta * k1byPi;
}

} // namespace ray_tracer
