#include "hip/hip_runtime.h"
﻿#include "translate.cuh"

namespace ray_tracer {

RT_DEVICE Translate::Translate(Hittable* hittable, const Vec3& offset) :
    hittable_(hittable), offset_(offset)
{
}

RT_DEVICE Translate::~Translate()
{
    delete hittable_;
}

RT_DEVICE bool Translate::hit(const Ray& ray, const float t_min, const float t_max, HitRecord& rec) const
{
    const Ray ray_displaced{ ray.origin() - offset_, ray.direction(), ray.time() };

    if (!hittable_->hit(ray_displaced, t_min, t_max, rec))
    {
        return false;
    }

    rec.hit_point += offset_;
    rec.set_face_normal(ray_displaced, rec.normal);

    return true;
}

RT_DEVICE bool Translate::bounding_box(const float ti, const float tf, AABB& box_out) const
{
    if (!hittable_->bounding_box(ti, tf, box_out))
    {
        return false;
    }

    box_out = AABB{ box_out.min() + offset_, box_out.max() + offset_ };
    return true;
}

} // namespace ray_tracer
