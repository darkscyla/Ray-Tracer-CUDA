#include "hip/hip_runtime.h"
﻿// --- Internal Includes ---
#include "xy_rect.cuh"
#include "solid_texture.cuh"

namespace ray_tracer {

RT_DEVICE XYRect::XYRect(const float x0, const float x1, const float y0,
    const float y1, const float z, Material* material) :
    x0_(x0), x1_(x1), y0_(y0), y1_(y1), z_(z),
    dx_inv_(1.0f / (x1 - x0)), dy_inv_(1.0f / (y1 - y0))
{
    material_ = material;
}

RT_DEVICE bool XYRect::hit(const Ray& ray, const float t_min, const float t_max, HitRecord& rec) const
{
    const auto t = (z_ - ray.origin().z()) / ray.direction().z();

    if (t < t_min || t > t_max)
    {
        return false;
    }

    const auto x = ray.origin().x() + t * ray.direction().x();
    const auto y = ray.origin().y() + t * ray.direction().y();

    if (x < x0_ || x > x1_ || y < y0_ || y > y1_)
    {
        return false;
    }

    rec.t = t;
    rec.hit_point = ray.at(t);
    const Vec3 outward_normal{ 0.0f, 0.0f, 1.0f };
    rec.set_face_normal(ray, outward_normal);
    rec.material = material_;

    rec.u = (x - x0_) * dx_inv_;
    rec.v = (y - y0_) * dy_inv_;

    return true;
}

RT_DEVICE bool XYRect::bounding_box(float ti, float tf, AABB& box_out) const
{
    box_out = AABB{ {x0_, y0_, z_ - 0.0001f}, {x1_, y1_, z_ + 0.0001f} };
    return true;
}

} // namespace ray_tracer
