﻿// --- Internal Includes ---
#include "xz_rect.cuh"
#include "solid_texture.cuh"

namespace ray_tracer {

RT_DEVICE XZRect::XZRect(const float x0, const float x1, const float z0,
    const float z1, const float y, Material* material) :
    x0_(x0), x1_(x1), z0_(z0), z1_(z1), y_(y),
    dx_inv_(1.0f / (x1 - x0)), dz_inv_(1.0f / (z1 - z0))
{
    material_ = material;
}

RT_DEVICE bool XZRect::hit(const Ray& ray, const float t_min, const float t_max, HitRecord& rec) const
{
    const auto t = (y_ - ray.origin().y()) / ray.direction().y();

    if (t < t_min || t > t_max)
    {
        return false;
    }

    const auto x = ray.origin().x() + t * ray.direction().x();
    const auto z = ray.origin().z() + t * ray.direction().z();

    if (x < x0_ || x > x1_ || z < z0_ || z > z1_)
    {
        return false;
    }

    rec.t = t;
    rec.hit_point = ray.at(t);
    const Vec3 outward_normal{ 0.0f, 1.0f, 0.0f };
    rec.set_face_normal(ray, outward_normal);
    rec.material = material_;

    rec.u = (x - x0_) * dx_inv_;
    rec.v = (z - z0_) * dz_inv_;

    return true;
}

RT_DEVICE bool XZRect::bounding_box(float ti, float tf, AABB& box_out) const
{
    box_out = AABB{ {x0_, y_ - 0.0001f, z0_}, {x1_, y_ + 0.0001f, z1_} };
    return true;
}

} // namespace ray_tracer
