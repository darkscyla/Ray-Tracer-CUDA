#include "hip/hip_runtime.h"
﻿// --- Internal Includes ---
#include "yz_rect.cuh"
#include "solid_texture.cuh"

namespace ray_tracer {

RT_DEVICE YZRect::YZRect(const float y0, const float y1, const float z0,
    const float z1, const float x, Material* material) :
    y0_(y0), y1_(y1), z0_(z0), z1_(z1), x_(x),
    dy_inv_(1.0f / (y1 - y0)), dz_inv_(1.0f / (z1 - z0))
{
    material_ = material;
}

RT_DEVICE bool YZRect::hit(const Ray& ray, const float t_min, const float t_max, HitRecord& rec) const
{
    const auto t = (x_ - ray.origin().x()) / ray.direction().x();

    if (t < t_min || t > t_max)
    {
        return false;
    }

    const auto y = ray.origin().y() + t * ray.direction().y();
    const auto z = ray.origin().z() + t * ray.direction().z();

    if (y < y0_ || y > y1_ || z < z0_ || z > z1_)
    {
        return false;
    }

    rec.t = t;
    rec.hit_point = ray.at(t);
    const Vec3 outward_normal{ 1.0f, 0.0f, 0.0f };
    rec.set_face_normal(ray, outward_normal);
    rec.material = material_;

    rec.u = (y - y0_) * dy_inv_;
    rec.v = (z - z0_) * dz_inv_;

    return true;
}

RT_DEVICE bool YZRect::bounding_box(float ti, float tf, AABB& box_out) const
{
    box_out = AABB{ {x_ - 0.0001f, y0_, z0_}, {x_ + 0.0001f, y1_, z1_} };
    return true;
}

} // namespace ray_tracer
